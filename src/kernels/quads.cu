#include "hip/hip_runtime.h"
#include <cmath>
#include <cassert>
#include <iostream>
#include "utils/cudaUtils.hpp"

namespace kernel {

	__global__ void countVisibleQuads ( 
			unsigned int *d_counter,
			unsigned char *d_voxel_grid,
			const unsigned int gridWidth, const unsigned int gridHeight, const unsigned int gridLength,
			const unsigned char threshold) {

		*d_counter = 0;
		__syncthreads();

		unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
		unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;
		unsigned int idz = blockIdx.z;
		unsigned int id = idz*gridWidth*gridHeight + idy*gridWidth + idx;


		if(idx >= gridWidth || idy >= gridHeight) {
			return;
		}

		__shared__ unsigned int localBlockCounter;
		localBlockCounter = 0;
		__syncthreads();

		unsigned char voxelValue = d_voxel_grid[id];	

		if(voxelValue > threshold) {
			unsigned char threadCounter = 0;
			threadCounter += (idx == 0 ? 1 : (d_voxel_grid[id - 1]<=threshold)); //left
			threadCounter += (idy == 0 ? 1 : (d_voxel_grid[id - gridWidth]<=threshold)); //down
			threadCounter += (idz == 0 ? 1 : (d_voxel_grid[id - gridHeight*gridWidth]<=threshold)); //back

			threadCounter += (idx == gridWidth  - 1 ? 1 : (d_voxel_grid[id + 1]<=threshold)); //right
			threadCounter += (idy == gridHeight - 1 ? 1 : (d_voxel_grid[id + gridWidth]<=threshold)); //up
			threadCounter += (idz == gridLength - 1 ? 1 : (d_voxel_grid[id + gridHeight*gridWidth]<=threshold)); //front

			atomicAdd(&localBlockCounter, threadCounter);
		}

		__syncthreads();
		if (threadIdx.x == 0 && threadIdx.y == 0) {
			atomicAdd(d_counter, localBlockCounter);
		}

	}



	__device__ __inline__ void writeVec3f(float *d_array, unsigned int arrayID, unsigned char stride, float x, float y, float z) {
#pragma unroll
		for(int i = 0; i < stride; i++) {
			d_array[3*stride*arrayID + 3*i + 0] = x;
			d_array[3*stride*arrayID + 3*i + 1] = y;
			d_array[3*stride*arrayID + 3*i + 2] = z;
		}
	}

	__global__ void 
		__launch_bounds__(1024)
		computeVisibleQuads( 
				unsigned int *d_counter,
				float *d_quads, float *d_normals, float *d_colors,
				unsigned char *d_voxel_grid,
				const unsigned int gridWidth, const unsigned int gridHeight, const unsigned int gridLength,
				const float cube_w, const float cube_h, const float cube_d,
				const unsigned char threshold,
				const unsigned char normalStride, const unsigned char colorStride) {


			unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
			unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;
			unsigned int idz = blockIdx.z % gridLength;
			unsigned int id = idz*gridWidth*gridHeight + idy*gridWidth + idx;


			if(idx >= gridWidth || idy >= gridHeight) {
				return;
			}

			unsigned char voxelValue = d_voxel_grid[id];	
			if (voxelValue <= threshold)
				return;

			unsigned char face = blockIdx.z/gridLength;
			bool draw = false;


			switch(face) {
				case(0): 
					{ //left
						draw = (idx == 0 ? true : (d_voxel_grid[id - 1] <= threshold)); //left
						break;
					}
				case(1): 
					{ //right
						draw =  (idx == gridWidth  - 1 ? true : (d_voxel_grid[id + 1] <= threshold)); //right
						break;
					}
				case(2): 
					{ //down
						draw = (idy == 0 ? true : (d_voxel_grid[id - gridWidth] <= threshold)); //down
						break;
					}
				case(3): 
					{ //up
						draw = (idy == gridHeight - 1 ? true : (d_voxel_grid[id + gridWidth] <= threshold)); //up
						break;
					}
				case(4): 
					{ //back
						draw = (idz == 0 ? true : (d_voxel_grid[id - gridHeight*gridWidth] <= threshold)); //back
						break;
					}
				case(5): 
					{ //front
						draw = (idz == gridLength - 1 ? true : (d_voxel_grid[id + gridHeight*gridWidth] <= threshold)); //front
						break;
					}
			}

			if(!draw)
				return;


			//get an array id
			const unsigned int arrayID = atomicAdd(d_counter, 1);

			//compute real position
			const float tx = idx*cube_w;
			const float ty = idy*cube_h;
			const float tz = idz*cube_d;

			/*write colors*/
			writeVec3f(d_colors, arrayID, colorStride, voxelValue/255.0f, voxelValue/255.0f, voxelValue/255.0f);

			switch(face) {
				case(0):
					{ //left
						writeVec3f(d_normals, arrayID, normalStride, -1.0f, 0.0f, 0.0f);
						writeVec3f(d_quads, 4*arrayID, 1, 0.0f + tx, 0.0f + ty, 0.0f + tz);
						writeVec3f(d_quads, 4*arrayID + 1, 1, 0.0f + tx, cube_h + ty, 0.0f + tz);
						writeVec3f(d_quads, 4*arrayID + 2, 1, 0.0f + tx, cube_h + ty, cube_d + tz);
						writeVec3f(d_quads, 4*arrayID + 3, 1, 0.0f + tx, 0.0f + ty, cube_d + tz);
						break;
					}
				case(1):
					{ //right
						writeVec3f(d_normals, arrayID, normalStride, +1.0f, 0.0f, 0.0f);
						writeVec3f(d_quads, 4*arrayID, 1, cube_w + tx, 0.0f + ty, 0.0f + tz);
						writeVec3f(d_quads, 4*arrayID + 1, 1, cube_w + tx, cube_h + ty, 0.0f + tz);
						writeVec3f(d_quads, 4*arrayID + 2, 1, cube_w + tx, cube_h + ty, cube_d + tz);
						writeVec3f(d_quads, 4*arrayID + 3, 1, cube_w + tx, 0.0f + ty, cube_d + tz);
						break;
					}
				case(2):
					{ //down
						writeVec3f(d_normals, arrayID, normalStride, 0.0f, -1.0f, 0.0f);
						writeVec3f(d_quads, 4*arrayID, 1, 0.0f + tx, 0.0f + ty, 0.0f + tz);
						writeVec3f(d_quads, 4*arrayID + 1, 1, 0.0f + tx, 0.0f + ty, cube_d + tz);
						writeVec3f(d_quads, 4*arrayID + 2, 1, cube_w + tx, 0.0f + ty, cube_d + tz);
						writeVec3f(d_quads, 4*arrayID + 3, 1, cube_w + tx, 0.0f + ty, 0.0f + tz);
						break;
					}
				case(3):
					{ //up
						writeVec3f(d_normals, arrayID, normalStride, 0.0f, 1.0f, 0.0f);
						writeVec3f(d_quads, 4*arrayID, 1, 0.0f + tx, cube_h + ty, 0.0f + tz);
						writeVec3f(d_quads, 4*arrayID + 1, 1, 0.0f + tx, cube_h + ty, cube_d + tz);
						writeVec3f(d_quads, 4*arrayID + 2, 1, cube_w + tx, cube_h + ty, cube_d + tz);
						writeVec3f(d_quads, 4*arrayID + 3, 1, cube_w + tx, cube_h + ty, 0.0f + tz);
						break;
					}
				case(4):
					{ //back
						writeVec3f(d_normals, arrayID, normalStride, 0.0f, 0.0f, -1.0f);
						writeVec3f(d_quads, 4*arrayID, 1, 0.0f + tx, 0.0f + ty, 0.0f + tz);
						writeVec3f(d_quads, 4*arrayID + 1, 1, cube_w + tx, 0.0f + ty, 0.0f + tz);
						writeVec3f(d_quads, 4*arrayID + 2, 1, cube_w + tx, cube_h + ty, 0.0f + tz);
						writeVec3f(d_quads, 4*arrayID + 3, 1, 0.0f + tx, cube_h + ty, 0.0f + tz);
						break;
					}
				case(5):
					{ //front
						writeVec3f(d_normals, arrayID, normalStride, 0.0f, 0.0f, 1.0f);
						writeVec3f(d_quads, 4*arrayID, 1, 0.0f + tx, 0.0f + ty, cube_d + tz);
						writeVec3f(d_quads, 4*arrayID + 1, 1, cube_w + tx, 0.0f + ty, cube_d + tz);
						writeVec3f(d_quads, 4*arrayID + 2, 1, cube_w + tx, cube_h + ty, cube_d + tz);
						writeVec3f(d_quads, 4*arrayID + 3, 1, 0.0f + tx, cube_h + ty, cube_d + tz);
						break;
					}
			}
		}

	
	void call_countVisibleQuads(dim3 dimGrid, dim3 dimBlock, unsigned int* nQuads_d, unsigned char *grid_d, unsigned int width, unsigned int height, unsigned int length, unsigned char threshold) {
			assert(nQuads_d);
			assert(grid_d);
			assert(width != 0);
			assert(height != 0);
			assert(length != 0);
			countVisibleQuads<<<dimGrid,dimBlock>>>(nQuads_d, grid_d, width, height, length, threshold);
			checkKernelExecution();
	}

}
