#include "hip/hip_runtime.h"
#include <cmath>

#include "utils/cudaUtils.hpp"
#include "utils/log.hpp"
#include "utils/utils.hpp"

namespace kernel {

	__global__ void 
	__launch_bounds__(512)
	cast(const long dataSize, float *float_data, unsigned char *char_data) {
		unsigned int id = blockIdx.y*65535*512 
			+ blockIdx.x*512 + threadIdx.x;

		if(id > dataSize)
			return;

		char_data[id] = (unsigned char) float_data[id];
	}

	void castKernel(unsigned long dataSize, float *float_data, unsigned char *char_data) {
		dim3 dimBlock(512);
		dim3 dimGrid((unsigned int)ceil(dataSize/512.0) % 65535, ceil(dataSize/(512*65535.0f)));
		log_console.infoStream() << "[KERNEL::Cast] <<<" << toStringDim(dimBlock) << ", " << toStringDim(dimGrid) << ">>>";
		cast<<<dimGrid,dimBlock>>>(dataSize, float_data, char_data);
	}


	__device__ unsigned short _atomicAddShort(unsigned short* address, unsigned short val) {
		unsigned int *base_address = (unsigned int *)((size_t)address & ~2);
		unsigned int old, assumed, sum, new_;

		old = *base_address;
		do {
			assumed = old;
			sum = val + (unsigned short)__byte_perm(old, 0, ((size_t)address & 2) ? 0x4432 : 0x4410);
			new_ = __byte_perm(old, sum, ((size_t)address & 2) ? 0x5410 : 0x3254);
			old = atomicCAS(base_address, assumed, new_);
		} while (assumed != old);

		return old;
	}

	__device__ unsigned char _atomicAddChar(unsigned char* address, unsigned char val) {

		unsigned int *base_address = (unsigned int *)((size_t)address & ~3);
		unsigned int selectors[] = {0x3214, 0x3240, 0x3410, 0x4210};
		unsigned int sel = selectors[(size_t)address & 3];
		unsigned int old, assumed, sum, new_;

		old = *base_address;
		do {
			assumed = old;
			sum = val + (unsigned short)__byte_perm(old, 0, ((size_t)address & 3));
			new_ = __byte_perm(old, sum, sel);
			old = atomicCAS(base_address, assumed, new_);
		} while (assumed != old);

		return old;
	}

	__global__ void 
	__launch_bounds__(512)
	computeMean(unsigned char *grid, unsigned short *hit_counter, unsigned int *sum, unsigned long nData) {
			unsigned int id = 512*65535*blockIdx.y + 512*blockIdx.x + threadIdx.x;  

			if(id >= nData)
				return;
			
			if(hit_counter[id] == 0u)
				grid[id] = 0u;
			else
				grid[id] = sum[id]/hit_counter[id];
		}
	
	void computeMeanKernel(unsigned char *grid, unsigned short *hit_counter, unsigned int *sum, 
			const unsigned long nData, hipStream_t stream) {
		dim3 dimBlock(512);
		dim3 dimGrid(((unsigned int)ceil(nData/512.0))%65535, ceil(nData/(512*65535.0f)));

		computeMean<<<dimGrid,dimBlock,0,stream>>>(grid, hit_counter, sum, nData);
	}


	__global__ void 
	__launch_bounds__(1024)
		VNN(const int nImages, const int imgWidth, const int imgHeight, 
				const float deltaGrid, const float deltaX, const float deltaY,
				const float xMin, const float yMin, const float zMin,
				const unsigned int gridIdx, const unsigned int gridIdy, const unsigned int gridIdz,
				const unsigned int voxelGridWidth, const unsigned int voxelGridHeight, const unsigned int voxelGridLength,
				float *offsetX, float *offsetY, float *offsetZ,
				float *r1, float *r2, float *r3, float *r4, float *r5, float *r6, float *r7, float *r8, float *r9,
				unsigned char *char_image_data, 
				unsigned char *voxel_data, 
				unsigned int *mean_grid,
				unsigned short *hit_counter) {

			unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;  
			unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y; 
			unsigned int id = idy*imgWidth + idx;
			unsigned int n = blockIdx.z;

			if(idx >= imgWidth || idy >= imgHeight)
				return;

			float vx = (idx+0.5f)*deltaX;	
			float vy = (idy+0.5f)*deltaY;

			float px = r1[n]*vx + r2[n]*vy + r3[n]*0.0f + offsetX[n] - xMin;
			float py = r4[n]*vx + r5[n]*vy + r6[n]*0.0f + offsetY[n] - yMin;
			float pz = r7[n]*vx + r8[n]*vy + r9[n]*0.0f + offsetZ[n] - zMin;

			unsigned int ix = __float2uint_rd(px/deltaGrid);
			unsigned int iy = __float2uint_rd(py/deltaGrid);
			unsigned int iz = __float2uint_rd(pz/deltaGrid);

			//check if the pixel is in the subgrid
			if(! (ix/voxelGridWidth == gridIdx 
						&& iy/voxelGridHeight == gridIdy
						&& iz/voxelGridLength == gridIdz)) {
				return;
			}

			ix %= voxelGridWidth;
			iy %= voxelGridHeight;
			iz %= voxelGridLength;
			unsigned long i = iz*voxelGridHeight*voxelGridWidth + iy*voxelGridWidth + ix;

			unsigned char value = char_image_data[id];

			_atomicAddShort(hit_counter + i, 1);	
			atomicAdd(mean_grid + i, (unsigned short)value);
		}

	void VNNKernel(const int nImages, const int imgWidth, const int imgHeight, 
			const float deltaGrid, const float deltaX, const float deltaY,
			const float xMin, const float yMin, const float zMin,
			const unsigned int gridIdx, const unsigned int gridIdy, const unsigned int gridIdz,
			const unsigned int voxelGridWidth, const unsigned int voxelGridHeight, const unsigned int voxelGridLength,
			float **offsets_d,
			float **rotations_d,
			unsigned char *char_image_data, 
			unsigned char *voxel_data, 
			unsigned int *mean_grid,
			unsigned short *hit_counter,
			hipStream_t stream) {

		dim3 dimBlock(32, 32, 1);
		dim3 dimGrid(ceil(imgWidth/32.0f), ceil(imgHeight/32.0f), nImages);

		log_console.infoStream() << "[KERNEL::VNN] <<<" << toStringDim(dimBlock) << ", " << toStringDim(dimGrid) << ", " << 0 << ", " << stream << ">>>";

		VNN<<<dimGrid,dimBlock,0,stream>>>(nImages, imgWidth, imgHeight, 
				deltaGrid,  deltaX,  deltaY,
				xMin, yMin, zMin,
				gridIdx, gridIdy, gridIdz,
				voxelGridWidth,  voxelGridHeight,  voxelGridLength,
				offsets_d[0], offsets_d[1], offsets_d[2],
				rotations_d[0], rotations_d[1], rotations_d[2], 
				rotations_d[3], rotations_d[4], rotations_d[5],
				rotations_d[6], rotations_d[7], rotations_d[8],
				char_image_data, voxel_data, mean_grid, hit_counter);

		checkKernelExecution();
	}

}

